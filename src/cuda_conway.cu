#include "hip/hip_runtime.h"
#include "cstdio"
#include "cuda_conway.h"
#include "conway_macros.h"
#include "scale.h"
#include "cuda_conway.cuh"

bool cuda_launch_conway(int shift, char** a, int numloops, int xmin, int xmax)
{
	if ((a == NULL) || (*a == NULL))
	{
		printf("NULL pointer passed\n");
		return false;
	}

	//size definitions
	int csize = BINARR_SIZE(CELL_SIZE(shift));
	int dimension = (1 << shift);
	int csizemem = (csize * sizeof(char));
	if((xmin == -1) || (xmax == -1))
	{
		xmin = 0;
		xmax = dimension;
	}

	//device problem distribution
	int blocksize = 128;
	int numblocks = (blocksize + dimension - 1)/dimension;
	
	char* device_source = NULL;
	char* device_destination = NULL;
	
	//allocate memory
	CHECKERR( hipMallocManaged(&device_source, csizemem) );
	CHECKERR( hipMallocManaged(&device_destination, csizemem) );
	
	//copy a into source
	CHECKERR( hipMemcpy(device_source, *a, csizemem, hipMemcpyHostToDevice) );
	
	for(int i = 0; i < numloops; ++i)
	{
		cuda_conway<<<numblocks,blocksize>>>(shift, device_source, device_destination, xmin, xmax);
		cuda_copyback<<<numblocks,blocksize>>>(shift, device_source, device_destination);
	}
	
	//copy result from device
	CHECKERR( hipMemcpy(*a, device_destination, csizemem, hipMemcpyDeviceToHost) );
	
	//free
	CHECKERR( hipFree(device_source) );
	CHECKERR( hipFree(device_destination) );
	
	return true;
}

__global__
void cuda_conway(int shift, char * a, char * b, int xmin, int xmax)
{
/*
	data will be partitioned by x value on MPI, and partitioned by y value on CUDA
	this will make memory accesses on the GPU more linear and less likely to hit a cache fault,
	resulting in probably unnoticeable speedup
*/
	int dimension = (1 << shift);
	int gridid = blockIdx.x * blockDim.x + threadIdx.x;
	int griddim = blockDim.x * gridDim.x;

	int ymin = ASSIGN_MIN(dimension, griddim, gridid);
	int ymax = ASSIGN_MAX(dimension, griddim, gridid);
	for(int y = ymin; y < ymax; ++y)
		for(int x = xmin; x < xmax; ++x)
			NEXT_CELL(x,y,shift,a,b);
}


__global__
void cuda_copyback(int shift, char * a, char * b)
{
	int dimension = (1 << shift);
	int gridid = blockIdx.x * blockDim.x + threadIdx.x;
	int griddim = blockDim.x * gridDim.x;

	int ymin = (ASSIGN_MIN(dimension, griddim, gridid) >> 3);
	int ymax = (ASSIGN_MAX(dimension, griddim, gridid) >> 3);

	int imin = ymin * dimension;
	int imax = ymax * dimension;
	for(int i = imin; i < imax; ++i)
		a[i] = b[i];
}

