#include "hip/hip_runtime.h"
#include "cstdio"
#include "cuda_conway.h"
#include "conway_macros.h"
#include "scale.h"
#include "cuda_conway.cuh"

bool cuda_launch_conway(int shift, char** a, char** b)
{
	if ((a == NULL) || (*a == NULL) || (b == NULL) || (*b == NULL))
	{
		printf("NULL pointer passed\n");
		return false;
	}
	
	//size constants
	int csize = BINARR_SIZE(CELL_SIZE(shift));
	int dimension = (1 << shift);
	int csizemem = (csize * sizeof(char));

	//device problem distribution
	int blocksize = 128;
	int numblocks = (blocksize + dimension - 1)/dimension;
	
	char* device_source = NULL;
	char* device_destination = NULL;
	
	//allocate memory
	CHECKERR( hipMallocManaged(&device_source, csizemem) );
	CHECKERR( hipMallocManaged(&device_destination, csizemem) );
	
	//copy a into source
	CHECKERR( hipMemcpy(device_source, *a, csizemem, hipMemcpyHostToDevice) );
	
	cuda_conway<<<numblocks,blocksize>>>(shift, device_source, device_destination);
	
	//copy result from device
	CHECKERR( hipMemcpy(*b, device_destination, csizemem, hipMemcpyDeviceToHost) );
	
	//free
	CHECKERR( hipFree(device_source) );
	CHECKERR( hipFree(device_destination) );
	
	return true;
}

__global__
void cuda_conway(int shift, char * a, char * b)
{
/*
	data will be partitioned by x value on MPI, and partitioned by y value on CUDA
	this will make memory accesses on the GPU more linear and less likely to hit a cache fault,
	resulting in probably unnoticeable speedup
*/
	int dimension = (1 << shift);
	int gridid = blockIdx.x * blockDim.x + threadIdx.x;
	int griddim = blockDim.x * gridDim.x;

	int ymin = ASSIGN_MIN(dimension, griddim, gridid);
	int ymax = ASSIGN_MAX(dimension, griddim, gridid);
	for(int y = ymin; y < ymax; ++y)
		for(int x = 0; x < dimension; ++x)
			NEXT_CELL(x,y,shift,a,b);
}
