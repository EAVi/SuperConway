#include "hip/hip_runtime.h"
#include "cstdio"
#include "cuda_conway.h"
#include "cuda_conway.cuh"

bool cuda_launch_conway(int shift, char** a, char** b)
{
	if (a == NULL || *a == NULL || b = NULL || *b = NULL)
	{
		printf("NULL pointer passed\n");
		return false;
	}
	
	//size constants
	int csize = CELL_SIZE(shift);
	int csizemem = (csize * sizeof(char));
	int width = (1 << shift);
	
	char* device_source = NULL;
	char* device_destination = NULL;
	
	//allocate memory
	CHECKERR( hipMallocManaged(&device_source, csizemem) );
	CHECKERR( hipMallocManaged(&device_destination, csizemem) );
	
	//copy a into source
	CHECKERR( hipMemcpy(device_source, *a, csizemem, hipMemcpyHostToDevice) );
	
	cuda_conway<<<1,1>>>(shift, device_source, device_destination);
	
	//copy result from device
	CHECKERR( hipMemcpy(*b, device_destination, csizemem, hipMemcpyDeviceToHost) );
	
	//free
	CHECKERR( hipFree(device_source) );
	CHECKERR( hipFree(device_destination) );
	
	return true;
}

__global__
void cuda_conway(int shift, char * a, char * b)
{
	int size = CELL_SIZE(shift);
	int dimension = (1 << shift);
	for(int y = 0; y < dimension; ++y)
		for(int x = 0; x < dimension; ++x)
			NEXT_CELL(x,y,shift,source,destination);
}